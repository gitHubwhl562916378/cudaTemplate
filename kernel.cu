#include <hip/hip_runtime.h>

#include <iostream>

__global__ void addArray(int *ary1, int *ary2)
{
    int indx = threadIdx.x;
    ary1[indx] = ary2[indx];
}

int main(int argc,char **argv)
{
    int ary[32]{0};
    int res[32]{0};

    for(int i = 0; i < 32; i++){
        ary[i] = 2*i;
    }
    int *d_ary, *d_res;
    hipMalloc((void**)&d_ary, 32 * sizeof(int));
    hipMalloc((void**)&d_res, 32 * sizeof(int));
    hipMemcpy((void*)d_ary, (void*)ary, 32 * sizeof(int), hipMemcpyHostToDevice);
    addArray<<<1,32>>>(d_res, d_ary);

    hipMemcpy((void*)res, (void*)d_res, 32 * sizeof(int), hipMemcpyDeviceToHost);

    for(int i = 0; i < 32; i++){
        std::cout << "result:" << res[i] << std::endl;
    }

    hipFree(d_ary);
    hipFree(d_res);
    return 0;
}
